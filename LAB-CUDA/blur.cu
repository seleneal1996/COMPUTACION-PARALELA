#include "hip/hip_runtime.h"
#include "lodepng.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"
#include <iostream>

#define BLUR_SIZE 7
#define R 0
#define G 1
#define B 2
#define A 3

__global__ void blurKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel, int channel, int copy_A) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if(col < width && row < height) {
    int pixVal = 0;
    int pixels = 0;
    if (copy_A)
      out[row*width*num_channel+col*num_channel+A] = in[row*width*num_channel+col*num_channel+A];
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
        int curRow = row + blurRow;
        int curCol = col + blurCol;
        if(curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
          pixVal += in[curRow * width * num_channel + curCol * num_channel + channel];
          pixels++;
        }
      }
    }
    out[row * width * num_channel + col * num_channel + channel] = (unsigned char)(pixVal/pixels);
  }
}

int main() {

  int width, height,n;
  unsigned char *image = stbi_load("image4.png",&width,&height,&n,0);
  unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
  unsigned char* Dev_Input_Image = NULL;
  unsigned char* Dev_Output_Image = NULL;
  hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
  hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);

  hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);

  //kernel call
  dim3 blockSize(16, 16, 1);
  dim3 gridSize(width/blockSize.x, height/blockSize.y,1);
  blurKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,R,0);
  blurKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,G,0);
  blurKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,B,1);
  
  hipDeviceSynchronize();

    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
  hipFree(Dev_Input_Image);
  hipFree(Dev_Output_Image);
  stbi_write_png("output_stbimage.png", width, height, n, image, width * n);


  return 0;
}